#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
//
//  main.cu
//  To compile the program:
//    nvcc main.cu -std=c++11 --expt-extended-lambda
//  Elegant Pairing Function
//
//  Created by Adolfo Solís on 4/4/19.
//  Copyright © 2019 Adolfo Solís. All rights reserved.
//

#include <iostream>
#include <math.h>
#include <omp.h>
#include <vector>
#define SIZE 10000
using namespace std;

struct RLE {
  int x;
  int y;
};

struct TIMER {
  float CPU;
  float GPU;
};

int elegantPair(int x, int y) {
  x = x >= 0 ? x * 2 : (x * -2) - 1;
  y = y >= 0 ? y * 2 : (y * -2) - 1;

  return (x >= y) ? (pow(x, 2) + x + y) : (pow(y, 2) + x);
}

vector<int> elegantUnpair(int z) {
  vector<int> tuple;

  int sqrtz = floor(sqrt(z));
  int sqz = sqrtz * sqrtz;

  if ((z - sqz) >= sqrtz) {
    tuple.push_back(sqrtz);
    tuple.push_back(z - sqz - sqrtz);
  } else {
    tuple.push_back(z - sqz);
    tuple.push_back(sqrtz);
  }

  tuple[0] = fmod(tuple[0], 2) == 0 ? tuple[0] / 2 : (tuple[0] + 1) / -2;
  tuple[1] = fmod(tuple[1], 2) == 0 ? tuple[1] / 2 : (tuple[1] + 1) / -2;

  return tuple;
}
// GPU Functions
thrust::device_vector<int> gpuEncoding(thrust::device_vector<RLE> rle) {
  thrust::device_vector<int> arrayCompressed(rle.size());

  // GPU - Elegant Pair Function
  auto gpuElegantPair = [=] __device__(RLE array) {
    int x = array.x;
    int y = array.y;
    x = x >= 0 ? x * 2 : (x * -2) - 1;
    y = y >= 0 ? y * 2 : (y * -2) - 1;

    return (x >= y) ? ((x * x) + x + y) : ((y * y) + x);
  };

  thrust::transform(rle.begin(), rle.end(), arrayCompressed.begin(),
                    gpuElegantPair);

  return arrayCompressed;
}

thrust::device_vector<RLE>
gpuDecoding(thrust::device_vector<int> arrayCompressed) {
  thrust::device_vector<RLE> rle(arrayCompressed.size());

  // GPU - Elegant Unpair Function
  auto gpuElegantUnpair = [=] __device__(int z) {
    RLE tuple;

    int sqrtz = floor(sqrt(z));
    int sqz = sqrtz * sqrtz;

    if ((z - sqz) >= sqrtz) {
      tuple.x = sqrtz;
      tuple.y = z - sqz - sqrtz;
    } else {
      tuple.x = z - sqz;
      tuple.y = sqrtz;
    }

    tuple.x = tuple.x % 2 == 0 ? tuple.x / 2 : (tuple.x + 1) / -2;
    tuple.y = tuple.y % 2 == 0 ? tuple.y / 2 : (tuple.y + 1) / -2;

    return tuple;
  };

  thrust::transform(arrayCompressed.begin(), arrayCompressed.end(), rle.begin(),
                    gpuElegantUnpair);

  return rle;
}

// CPU Functions
vector<int> cpuEncode(int *rle_1, int *rle_2, int size) {
  vector<int> arrayCompressed;
  for (int index = 0; index < size; index++) {
    arrayCompressed.push_back(elegantPair(rle_1[index], rle_2[index]));
  }
  return arrayCompressed;
}

vector<vector<int>> cpuDecode(vector<int> arrayCompressed) {
  vector<vector<int>> rle;
  for (int index = 0; index < arrayCompressed.size(); index++) {
    vector<int> tuple = elegantUnpair(arrayCompressed[index]);
    rle.push_back(tuple);
  }
  return rle;
}

float differentElements(vector<vector<int>> CPU, thrust::host_vector<RLE> GPU) {
  int diff = 0;
  for (int i = 0; i < CPU.size(); ++i) {
    if (CPU[i][0] != GPU[i].x || CPU[i][1] != GPU[i].y)
      ++diff;
  }
  return diff * 100 / CPU.size();
}

TIMER differenceExecTime(float cpu, float gpu) {
  TIMER result;
  float mayor = cpu > gpu ? cpu : gpu;
  float menor = cpu < gpu ? cpu : gpu;

  result.CPU = 100 - (menor * 100 / mayor);
  result.GPU = 100 - (menor * 100 / mayor);

  return result;
}

void whoWins(TIMER times) {
  if (times.CPU > times.GPU)
    cout << "CPU WINS! : " << times.CPU << "%%" << endl;
  else
    cout << "GPU WINS! : " << times.GPU << "%%" << endl;
}

int main(int argc, const char *argv[]) {
  hipSetDevice(1);
  TIMER timer;
  srand((int)time(NULL));

  // Initialize the cuda timers
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;

  int *rle_1;
  int *rle_2;

  rle_1 = (int *)malloc(sizeof(int) * SIZE);
  rle_2 = (int *)malloc(sizeof(int) * SIZE);

  // Generate CPU array of size 'SIZE'
  thrust::host_vector<RLE> rle(SIZE);

  // Initialize Vectors CPU
  for (int i = 0; i < SIZE; i++) {
    rle[i].x = rle_1[i] = rand() % 100;
    rle[i].y = rle_2[i] = rand() % 100;
  }

  // Copy CPU vectors to GPU
  thrust::device_vector<RLE> d_rle = rle;

  // Compress on GPU
  cout << "Compressing GPU.." << endl;
  hipEventRecord(start);
  thrust::device_vector<int> arrayCompressedDevice = gpuEncoding(d_rle);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  float timer_compress_gpu = milliseconds;
  cout << "GPU time compress: " << milliseconds << endl;
  // Copy GPU vectors to CPU
  thrust::host_vector<int> arrayCompressedHost = arrayCompressedDevice;

  // for (int i = 0; i < arrayCompressedHost.size(); i++) {
  //   cout << arrayCompressedHost[i] << endl;
  // }

  // // Decompress on GPU
  cout << "Decompressing GPU.." << endl;
  hipEventRecord(start);
  thrust::device_vector<RLE> res_rle_gpu = gpuDecoding(arrayCompressedDevice);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  float timer_decompress_gpu = milliseconds;
  cout << "GPU time decompress: " << milliseconds << endl;

  // Copy GPU vectors to CPU
  thrust::host_vector<RLE> arrayDecompressedHost = res_rle_gpu;

  // for (int i = 0; i < arrayDecompressedHost.size(); i++) {
  //   cout << arrayDecompressedHost[i].x << ", " << arrayDecompressedHost[i].y
  //        << endl;
  // }

  // Compress on CPU
  cout << "Compressing CPU.." << endl;
  hipEventRecord(start);
  vector<int> arrayCompressed = cpuEncode(rle_1, rle_2, SIZE);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  float timer_compress_cpu = milliseconds;
  cout << "CPU time compress: " << milliseconds << endl;

  // for (int index = 0; index < arrayCompressed.size(); index++) {
  //   cout << arrayCompressed[index] << endl;
  // }

  cout << "Decompressing CPU.." << endl;
  hipEventRecord(start);
  vector<vector<int>> res_rle = cpuDecode(arrayCompressed);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  float timer_decompress_cpu = milliseconds;
  cout << "CPU time decompress: " << milliseconds << endl;

  // for (int index = 0; index < res_rle.size(); index++) {
  //   cout << res_rle[index][0] << ", " << res_rle[index][1] << endl;
  // }

  cout << "Percentage of different elements: "
       << differentElements(res_rle, arrayDecompressedHost) << endl;

  cout << "Compression CPU vs GPU..." << endl;
  timer = differenceExecTime(timer_compress_cpu, timer_compress_gpu);
  whoWins(timer);

  cout << "Decompression CPU vs GPU..." << endl;
  timer = differenceExecTime(timer_decompress_cpu, timer_decompress_gpu);
  whoWins(timer);

  free(rle_1);
  free(rle_2);

  return 0;
}
